#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <sys/time.h>
#include <cstdlib> 
#include <fstream>
#include <iostream>
#include <string>
#include <unistd.h>
#include "apis_cu.h"
#include "hip/hip_runtime.h"
#include ""
#define BLOCK_DIM 10

__global__ void matrix_mul_gpu(int64_t *M, int64_t *N, int64_t *P, int64_t widthA, int64_t heightA,
                               int64_t widthB) {
    int64_t i = threadIdx.x + blockDim.x * blockIdx.x;
    int64_t j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < widthB && j < heightA) {
        int64_t sum = 0;
        for (int64_t k = 0; k < widthA; k++) {
            int64_t a = M[j * widthA + k];
            int64_t b = N[k * widthB + i];
            sum += a * b;
        }
        P[j * widthB + i] = sum;
    }
}

int Row_A = 0, Col_A = 0, Row_B = 0, Col_B = 0;
int main(int argc, char **argv) {
    // 读取本进程所代表的chiplet编号
    int srcX = atoi(argv[1]);
    int srcY = atoi(argv[2]);
    while (1) {
        int64_t *size_A = new int64_t[2];
        int64_t *size_B = new int64_t[2];
        int64_t *Size_A, *Size_B;
        hipMalloc((void **)&Size_A, sizeof(int64_t) * 2);
        hipMalloc((void **)&Size_B, sizeof(int64_t) * 2);

        receiveMessage(srcX, srcY, 0, 0, Size_A, sizeof(int64_t) * 2);
        receiveMessage(srcX, srcY, 0, 0, Size_B, sizeof(int64_t) * 2);

        hipMemcpy(size_A, Size_A, sizeof(int64_t) * 2, hipMemcpyDeviceToHost);
        hipMemcpy(size_B, Size_B, sizeof(int64_t) * 2, hipMemcpyDeviceToHost);
        Row_A = size_A[0];
        Col_A = size_A[1];
        Row_B = size_B[0];
        Col_B = size_B[1];
        if(Row_A == -1 && Col_A == -1 && Row_B == -1 && Col_B == -1){
            hipFree(Size_A);
            hipFree(Size_B);

            delete[] size_A;
            delete[] size_B;
            break;
        }
        int64_t *C = (int64_t *)malloc(sizeof(int64_t) * Col_B * Row_A);
        int64_t *A = (int64_t *)malloc(sizeof(int64_t) * Row_A * Col_A);

        int64_t *d_dataA, *d_dataB, *d_dataC;
        hipMalloc((void **)&d_dataA, sizeof(int64_t) * Row_A * Col_A);
        hipMalloc((void **)&d_dataB, sizeof(int64_t) * Row_B * Col_B);
        hipMalloc((void **)&d_dataC, sizeof(int64_t) * Col_B * Row_A);

        receiveMessage(srcX, srcY, 0, 0, d_dataA, Col_A * Row_A * sizeof(int64_t));
        receiveMessage(srcX, srcY, 0, 0, d_dataB, Col_B * Row_B * sizeof(int64_t));

        hipMemcpy(A, d_dataA, sizeof(int64_t) * Col_A * Row_A, hipMemcpyDeviceToHost);
        for (int64_t i = 0; i < Row_A * Col_A; i++) {
            std::cout << A[i];
            if (i % Col_A == 0 && i != 0)
                std::cout << std::endl;
            else
                std::cout << " ";
        }
        // calculate
        dim3 threadPerBlock(BLOCK_DIM, BLOCK_DIM);
        // dim3 blockNumber(1);
        dim3 blockNumber((Col_B + threadPerBlock.x - 1) / threadPerBlock.x,
                         (Row_A + threadPerBlock.y - 1) / threadPerBlock.y);
        matrix_mul_gpu<<<blockNumber, threadPerBlock>>>(d_dataA, d_dataB, d_dataC, Col_A, Row_A,
                                                        Col_B);
        hipMemcpy(C, d_dataC, sizeof(int64_t) * Row_A * Col_B, hipMemcpyDeviceToHost);
        for (int64_t i = 0; i < Row_A * Col_B; i++) {
            std::cout << C[i];
            if (i % Col_B == 0 && i != 0)
                std::cout << std::endl;
            else
                std::cout << " ";
        }
        sendMessage(0, 0, srcX, srcY, d_dataC, Row_A * Col_B * sizeof(int64_t));
        hipFree(d_dataA);
        hipFree(d_dataB);
        hipFree(d_dataC);
    }
    return 0;
}